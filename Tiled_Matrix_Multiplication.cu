#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

// Assuming a practical block size that fits within shared memory limits
const int TILE_WIDTH = 16;

__global__ void MatrixMulKernel(int *A, int *B, int *C, int N, int K, int M) {
    int bx = blockIdx.x, by = blockIdx.y;
    int tx = threadIdx.x, ty = threadIdx.y;
    int Row = by * TILE_WIDTH + ty;
    int Col = bx * TILE_WIDTH + tx;

    __shared__ int s_A[TILE_WIDTH][TILE_WIDTH];
    __shared__ int s_B[TILE_WIDTH][TILE_WIDTH];

    int Cvalue = 0;
    for (int m = 0; m < (K-1)/TILE_WIDTH + 1; ++m) {
        if (Row < N && m*TILE_WIDTH + tx < K)
            s_A[ty][tx] = A[Row*K + m*TILE_WIDTH + tx];
        else
            s_A[ty][tx] = 0;

        if (Col < M && m*TILE_WIDTH + ty < K)
            s_B[ty][tx] = B[(m*TILE_WIDTH + ty)*M + Col];
        else
            s_B[ty][tx] = 0;

        __syncthreads();

        for (int k = 0; k < TILE_WIDTH; ++k)
            Cvalue += s_A[ty][k] * s_B[k][tx];

        __syncthreads();
    }

    if (Row < N && Col < M)
        C[Row*M + Col] = Cvalue;
}

double getTime() {
  struct timeval tv;
  gettimeofday(&tv, NULL);
  return (double)tv.tv_sec + (double)tv.tv_usec * 1e-6;
}

int main() {
    int N = 1000; // Number of rows in A and C
    int K = 2000; // Number of columns in A and rows in B
    int M = 1000; // Number of columns in B and C

    size_t sizeA = N * K * sizeof(int);
    size_t sizeB = K * M * sizeof(int);
    size_t sizeC = N * M * sizeof(int);

    int *h_a, *h_b, *h_c;
    int *d_a, *d_b, *d_c;

    // Allocate host memory
    h_a = (int *)malloc(sizeA);
    h_b = (int *)malloc(sizeB);
    h_c = (int *)malloc(sizeC);

    // Initialize matrices
    for (int i = 0; i < N * K; i++) h_a[i] = rand() % 100;
    for (int i = 0; i < K * M; i++) h_b[i] = rand() % 100;

    // Allocate device memory
    hipMalloc(&d_a, sizeA);
    hipMalloc(&d_b, sizeB);
    hipMalloc(&d_c, sizeC);

    // Copy matrices to the device
    hipMemcpy(d_a, h_a, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, sizeB, hipMemcpyHostToDevice);

    // Determine the number of threads and blocks
    dim3 threadsPerBlock(TILE_WIDTH, TILE_WIDTH);
    dim3 blocksPerGrid(ceil((M + TILE_WIDTH - 1) / TILE_WIDTH), ceil((N + TILE_WIDTH - 1) / TILE_WIDTH));

    double start = getTime();
    MatrixMulKernel<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, N, K, M);
    hipDeviceSynchronize(); // Wait for GPU to finish
    double end = getTime();

    // Copy the result back to host
    hipMemcpy(h_c, d_c, sizeC, hipMemcpyDeviceToHost);

    // Print some results
    printf("Result Matrix:\n");
    for (int i = 0; i < 10; i++) {
        for (int j = 0; j < 10; j++) {
            printf("%d ", h_c[i * M + j]);
        }
        printf("\n");
    }

    printf("Execution Time: %.6f seconds\n", end - start);

    // Cleanup
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}
