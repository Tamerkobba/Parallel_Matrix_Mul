
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
 const int SHMEM_SIZE = 1 << 10;
__global__ void MatrixMulKernel(int *a, int *b, int *c, int N) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  // Statically allocated shared memory
  __shared__ int s_a[SHMEM_SIZE];
  __shared__ int s_b[SHMEM_SIZE];
 int tmp = 0;

  // Sweep tile across matrix
  for (int i = 0; i < N; i += blockDim.x) {
    // Load in elements for this tile
    s_a[threadIdx.y * blockDim.x + threadIdx.x] = a[row * N + i + threadIdx.x];
    s_b[threadIdx.y * blockDim.x + threadIdx.x] =
        b[i * N + threadIdx.y * N + col];

    // Wait for both tiles to be loaded in before doing computation
    __syncthreads();

    // Do matrix multiplication on the small matrix
    for (int j = 0; j < blockDim.x; j++) {
      tmp +=
          s_a[threadIdx.y * blockDim.x + j] * s_b[j * blockDim.x + threadIdx.x];
    }

    // Wait for all threads to finish using current tiles before loading in new
    // ones
    __syncthreads();
  }
    // Write back results
  c[row * N + col] = tmp;}

// CUDA Kernel function to multiply the matrices is assumed to be defined elsewhere

double getTime() {
  struct timeval tv;
  gettimeofday(&tv, NULL);
  return (double)tv.tv_sec + (double)tv.tv_usec * 1e-6;
}

int main() {
  int N = 1 << 10;
  size_t size = N * N * sizeof(int);

  // host copies of matrices a, b & c
  int *h_a = (int *)malloc(size);
  int *h_b = (int *)malloc(size);
  int *h_c = (int *)malloc(size);
  int *d_a, *d_b, *d_c; // device copies of a, b, & c

  // Setup input values
  for (int i = 0; i < N * N; i++) {
    h_a[i] = rand() % 100;
    h_b[i] = rand() % 100;
  }

  // Allocate space for device copies of a, b, c
  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);

  // Copy inputs to device
  hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

  // Launch add() kernel on GPU with N blocks
  dim3 threadsPerBlock(16, 16);
  dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x, (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

  double start = getTime();
  MatrixMulKernel<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, N);
  hipDeviceSynchronize(); // Wait for the GPU to finish
  double end = getTime();

  // Copy result back to host
  hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

  // Clean up
  hipFree(d_a); hipFree(d_b); hipFree(d_c);

  printf("Result Matrix:\n");
  // Print a subset of the result matrix
  for (int i = 0; i < 10; i++) {
    for (int j = 0; j < 10; j++) {
      printf("%d ", h_c[i * N + j]);
    }
    printf("\n");
  }

  printf("Execution Time: %.6f seconds\n", end - start);

  free(h_a); free(h_b); free(h_c);

  return 0;
}
